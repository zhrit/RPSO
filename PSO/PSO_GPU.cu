#include "hip/hip_runtime.h"
/***************************************************************
*  ���ļ���������Ⱥ�㷨���й���GPU�ĺ����Ķ���                        *
*  @author   ZhRiT                                             *
*  @email    buaazhangrit@163.com                              *
*  @version  1.0.1                                             *
*  @date     2018-10-31                                        *
***************************************************************/
#include "pch.h"
#include "PSO.cuh"
#include <hip/hip_runtime.h>
#include <time.h>
#include "psocudalib.cuh"
#include <iostream>
#include <>

using namespace std;
/**
 * @brief ��CPU������
 */
void PSO::Run_GPU() {
	cout << "GPU" << endl;

	// ��ʼ��cuda
	if (!psocudalib::InitCUDA()) {
		cout << "CUDA initializa failed!" << endl;
		return;
	}

	// �ڴ��С
	unsigned int memoriaSize1 = (m_d * m_number) * sizeof(double);
	unsigned int memoriaSize2 = (m_number) * sizeof(double);
	unsigned int memoriaSize3 = (m_d) * sizeof(double);

	double *xx;       // λ��
	double *vx;       // �ٶ�
	double *value;    // Ŀ�꺯��ֵ
	double *pbestx;   // ��������λ��
	double *pbest;    // ��������ֵ
	double *min;      // λ���½�
	double *max;      // λ���Ͻ�
	int *gbest;       // ����λ�õı��?

	// �����ڴ�
	hipMalloc((void**)&xx, memoriaSize1);
	hipMalloc((void**)&vx, memoriaSize1);
	hipMalloc((void**)&pbestx, memoriaSize1);
	hipMalloc((void**)&pbest, memoriaSize2);
	hipMalloc((void**)&max, memoriaSize3);
	hipMalloc((void**)&min, memoriaSize3);

	hipMalloc((void**)&value, memoriaSize2);//?
	hipMalloc((void**)&gbest, sizeof(int));//?

	// ���ݴ���CPU->GPU
	hipMemcpy(min, m_min, memoriaSize3, hipMemcpyHostToDevice);
	hipMemcpy(max, m_max, memoriaSize3, hipMemcpyHostToDevice);

	dim3 threads(16, 16);
	dim3 blocks(2, 2);

	dim3 threadsN(6, 6);
	dim3 blocksN(1, 1);
	// ��ʼ������
	psokernel::InitParticles << <blocks, threads >> > (xx, vx, pbestx, gbest, m_d, m_number, min, max);

	int iter = 0;
	do {
		psokernel::GetFitness << <blocksN, threadsN >> > (xx, value, m_d, m_number, 1); //����Ŀ�꺯��ֵ
		iter++;
	} while (iter < 1); //m_tMax

	m_t_act = iter;

	double *rxx = new double[m_d * m_number];
	double *rvx = new double[m_d * m_number];
	double *rv = new double[m_number];

	hipMemcpy(rxx, xx, memoriaSize1, hipMemcpyDeviceToHost);
	hipMemcpy(rvx, vx, memoriaSize1, hipMemcpyDeviceToHost);
	hipMemcpy(rv, value, memoriaSize2, hipMemcpyDeviceToHost);

	cout << "x: ";
	for (int i = 0; i < m_d * m_number; i++) {
		if (i % m_d == 0) {
			cout << "\n";
		}
		cout << rxx[i] << ", ";
	}
	cout << "\nv: ";
	for (int i = 0; i < m_d * m_number; i++) {
		if (i % m_d == 0) {
			cout << "\n";
		}
		cout << rvx[i] << ", ";
	}
	cout << "\nvalue: ";
	for (int i = 0; i < m_number; i++) {
		cout << rv[i] << ", \n";
	}
	return;
}


namespace psokernel {
	/**
	 * @breif �����ʼ������
	 * @param xx      λ��
	 * @param vx      �ٶ�
	 * @param pbestx  ��������λ��
	 * @param gbest   ȫ�����ű��
	 * @param d       ά��
	 * @param n       ���Ӹ���
	 * @param min     λ���½�
	 * @param max     λ���Ͻ�
	 */
	__global__ void InitParticles(double *xx, double *vx, double *pbestx, int *gbest, int d, int n, double *min, double *max) {
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;
		// ������
		int index = bx * gridDim.x * blockDim.x * blockDim.y + by * blockDim.x * blockDim.y + ty * blockDim.x + tx;

		if (index < d * n) {
			int dind = index % d;
			xx[index] = (max[dind] - min[dind]) * psocudalib::randomNumber(index) + min[dind];
			pbestx[index] = xx[index];

			vx[index] = (max[dind] - min[dind]) * psocudalib::randomNumber(index) + min[dind] - xx[index];

			if (index == 0) *gbest = 0;
		}
	}

	/**
	 * @breif ��Ŀ�꺯��ֵ
	 * @param xx      λ��
	 * @param value   Ŀ�꺯��ֵ
	 * @param d       ά��
	 * @param n       ���Ӹ���
	 * @param funcIdx Ŀ�꺯�����
	 */
	__global__ void GetFitness(double *xx, double *value, int d, int n, int funcIdx) {
		int tx = threadIdx.x;
		int ty = threadIdx.y;
		int index = ty * blockDim.x + tx; // �������
		int offset = index * d; // ������xx�����е�ƫ�ƣ�λ�ã�

		if (index < n) {
			double val = 0.0;
			val = psocudalib::sphere(xx, d, offset);

			value[index] = val;
		}
	}
}
